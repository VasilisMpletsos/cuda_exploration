
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    
    // Get the limit of the current cuda support
    int device;
    hipDeviceProp_t props;
    hipGetDevice(&device);
    hipGetDeviceProperties(&props, device);
    int threads_per_block = props.maxThreadsPerBlock;
    printf("Maximum threads per block: %d\n", threads_per_block);

    return 0;
}